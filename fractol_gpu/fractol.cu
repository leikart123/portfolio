extern "C"{
    #include "fractol.h"
    }
    #include "fractol.cuh"

    __host__ int main(int argc, char **argv)
    {
        c_cntrl	*cntrl;

        if (!(cntrl = (c_cntrl *)malloc(sizeof(c_cntrl))))
            return (0);
        mlx(cntrl);
        if (args_parse(argc, argv, cntrl) < 0)
            exit(0);
        hipMalloc((int **)&cntrl->d_data_ptr, sizeof(int) *NumberOfPixel);
        draw_call(cntrl);
        key_mouse_control(cntrl);
        mlx_loop(cntrl->mlx);
        return (0);
    }