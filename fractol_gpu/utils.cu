extern "C"{
    #include "fractol.h"
    }
    #include "fractol.cuh"

__host__ void    draw_call(c_cntrl *cntrl)
{
    dim3     gridSize;
    dim3     blockSize;
    double minx;
    double miny;

    minx = (cntrl->x_pos + (WID >> 1)) / (cntrl->zoom / 2) / -2;
    miny = (cntrl->y_pos + (WID >> 1)) / (cntrl->zoom / 2) / -2;
    gridSize = dim3(1, HEI, 1);
    blockSize = dim3(threadNum, 1, 1);
    if(cntrl->type == JULIA)
        draw_julia<<<gridSize, blockSize>>>(WID, HEI, cntrl->zoom, cntrl->d_data_ptr, minx, miny, cntrl->x_pos, cntrl->y_pos);
    if(cntrl->type == MANDELBROT)
        draw_mandelbrot<<<gridSize, blockSize>>>(WID, HEI, cntrl->zoom, cntrl->d_data_ptr, minx, miny, cntrl->x_pos, cntrl->y_pos);
    hipMemcpy(cntrl->h_data_ptr, cntrl->d_data_ptr, sizeof(int) * NumberOfPixel, hipMemcpyDeviceToHost);
    mlx_put_image_to_window(cntrl->mlx, cntrl->win, cntrl->img, 0, 0);    
}