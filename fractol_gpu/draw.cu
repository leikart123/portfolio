
#include <hip/hip_runtime.h>
__global__ void draw_julia(int w, int h, double zoom, int *data_ptr, double minx, double miny, double posX, double posY)
{
    double cRe, cIm;
    double newRe, newIm, oldRe, oldIm;
    int maxIterations = 880000;
    cRe = -0.74543;
    cIm = 0.11301;
    int r;
    int g;
    int b;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    if (x >= w || y>= h)
        return; 
    newRe = x / zoom + minx + posX;
    newIm = y / zoom + miny + posY;
    int i;
    for (i = 0; i < maxIterations; i++)
    {
        oldRe = newRe;
        oldIm = newIm;
        newRe = oldRe * oldRe - oldIm * oldIm + cRe;
        newIm = 2 * oldRe * oldIm + cIm;
        if ((newRe * newRe + newIm * newIm) > 2) break;
    }
        r = (i * 15) % 255;
        g = (i * 6) % 255;
        b = (i * 9) % 255;
        data_ptr[y * h + x] = ((r << 16) | (g << 8) | b);
}

__global__ void draw_mandelbrot(int w, int h, double zoom, int *data_ptr, double minx, double miny, double posX, double posY)
{
    double z_re;
    double z_im;
    double c_re;
    double c_im;
    double new_z_re;
    double new_z_im;
    int maxIterations = 1000;
    int r;
    int g;
    int b;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int i;
    if (x >= w || y>= h)
        return;
    c_re = x / zoom + minx + posX;
    c_im = y / zoom + miny + posY;
    z_re = 0;
    z_im = 0;
    for(i = 0; i < maxIterations; i++)
    {
        new_z_re = z_re * z_re;
		new_z_im = z_im * z_im;
		if (new_z_re + new_z_im > 4)
			break ;
		z_im = 2 * z_re * z_im + c_im;
		z_re = new_z_re - new_z_im + c_re;
    }
    r = (i * 15) % 255;
    g = (i * 6) % 255;
    b = (i * 9) % 255;
    data_ptr[y * h + x] = ((r << 16) | (g << 8) | b);
}